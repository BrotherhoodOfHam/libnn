#include "hip/hip_runtime.h"
/*
    Random number generator
*/

#include <random>

#include "kernels.h"
#include "nn/device.h"

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

using namespace nn;

/*************************************************************************************************************************************/

__global__ void RNG_bernoulli_kernel(float* x, uint n, random_generator::seed_type seed, float p)
{
    uint i = global_index();
    if (i < n)
    {
        hiprandState local_state;
        hiprand_init(seed, threadIdx.x, 0, &local_state);

        x[i] = (hiprand_uniform(&local_state) < p) ? 1.0f : 0.0f;
    }
}

/*************************************************************************************************************************************/

random_generator::random_generator() :
    random_generator(std::random_device()())
{}

random_generator::~random_generator()
{
    check(hiprandDestroyGenerator(_prng));
}

random_generator::random_generator(seed_type seed)
{
    check(hiprandCreateGenerator(&_prng, HIPRAND_RNG_PSEUDO_DEFAULT));
    check(hiprandSetPseudoRandomGeneratorSeed(_prng, seed));
}

/*
void random_generator::init(seed_type seed, size_t size)
{
    bool reseed = seed != _seed;
    if (size > _states_size)
    {
        _allocator.free();
        _states = _allocator.alloc_array<hiprandState>(size);
        _states_size = size;
        reseed = true;
    }

    if (reseed)
    {
        _seed = seed;

        uint block_size = 256;
        uint block_count = ((uint)size + block_size - 1) / block_size;
        RNG_init_kernel<<<block_count, block_size>>>((hiprandState*)_states, _seed, (uint)size);
    }
}
*/

void random_generator::seed(seed_type seed)
{
    check(hiprandSetPseudoRandomGeneratorSeed(_prng, seed));
}

void random_generator::random_uniform(vector x)
{
    check(hiprandGenerateUniform(_prng, x.ptr(), x.size()));
}

void random_generator::random_normal(vector x, float sdv, float mean)
{
    check(hiprandGenerateNormal(_prng, x.ptr(), x.size(), mean, sdv));
}

void random_generator::random_bernoulli(vector x, float probability)
{
    uint block_size = 256;
    uint block_count = ((uint)x.size() + block_size - 1) / block_size;

    //curandGenerateBinomial(_prng, x.ptr(), x.size(), 1u, (double)probability);

    seed_type seed = std::random_device()();
    RNG_bernoulli_kernel<<<block_count, block_size>>>(x.ptr(), x.size(), seed, probability);
}

/*************************************************************************************************************************************/
